/*
 * DHMProcessor.cu
 *
 *  Created on: Jun 14, 2017
 *      Author: michaelmurphy
 */

#include <opencv2/opencv.hpp>
#include <opencv2/highgui.hpp>

#include <boost/range.hpp>
#include <boost/filesystem.hpp>

#include "DHMCommon.cuh"
#include "DHMProcessor.cuh"

//namespace YipLab {

DHMProcessor::DHMProcessor(std::string inputDir, std::string outputDir) {
    this->inputDir = inputDir;
    this->outputDir = outputDir;

    // reset the GPU, use proper exceptions to do this...
    CUDA_CHECK( hipDeviceReset() );

    // camera crap would go here...

    // make sure input, output directories are fine
    using namespace boost::filesystem;
    if ( !is_directory(inputDir) ) throw DHMException("Input directory not found", __LINE__, __FILE__);
    if ( !is_directory(outputDir) ) throw DHMException("Output directory not found", __LINE__, __FILE__);

    // pack parameters
    p = { N, NUM_SLICES, NUM_FRAMES, DX, DY, DZ, Z0, LAMBDA0 };

    // allocate buffers, setup FFT

    // what a fucking mess
//        complex *image;
//        checkCudaErrors( hipMalloc(&image, N*N*sizeof(complex)) );
//        complex *psf;
//        checkCudaErrors( hipMalloc(&psf, N*N*sizeof(complex)) );
//
//        complex *host_psf;
//        checkCudaErrors( hipHostMalloc(&host_psf, NUM_SLICES*(N/2+1)*(N/2+1)*sizeof(complex)) );
//
//        byte *image_u8;
//        checkCudaErrors( hipMalloc(&image_u8, N*N*sizeof(byte)) );
//
//        hipStream_t math_stream, copy_stream;
//        checkCudaErrors( hipStreamCreate(&math_stream) );
//        checkCudaErrors( hipStreamCreate(&copy_stream) );
//
//        complex *in_buffers[2];
//        checkCudaErrors( hipMalloc(&in_buffers[0], NUM_SLICES*N*N*sizeof(complex)) );
//        checkCudaErrors( hipMalloc(&in_buffers[1], NUM_SLICES*N*N*sizeof(complex)) );
//
//        real *out_buffer;
//        checkCudaErrors( hipMalloc(&out_buffer, NUM_SLICES*N*N*sizeof(real)) );
//
//        // managed memory would be much nicer here, esp on Tegra, but was causing problems w/ streams
//        char *host_mask, *mask;
//        // checkCudaErrors( hipMallocManaged(&mask, NUM_SLICES*sizeof(char), hipMemAttachGlobal) );
//        checkCudaErrors( hipHostMalloc(&host_mask, NUM_SLICES*sizeof(char)) );
//        checkCudaErrors( hipMalloc(&mask, NUM_SLICES*sizeof(char)) );
//        memset(host_mask, 1, NUM_SLICES);
//        checkCudaErrors( hipMemcpy(mask, host_mask, NUM_SLICES*sizeof(char), hipMemcpyHostToDevice) );

}

void DHMProcessor::process_camera() {
    // stub
    // also needs to save images
}

void DHMProcessor::process_folder() {
    using namespace boost::filesystem;

    float *volume = new float[NUM_SLICES*N*N];
//        const int vol_sizes[3] = {NUM_SLICES, N, N};

    for (auto &f : boost::make_iterator_range(directory_iterator(inputDir), {}))
    {
        std::string path = f.path().string();

        // make sure the image is NxN? make sure it's an image / bitmap?
        if (path.substr(path.find_last_of(".") + 1) != "bmp")
            throw DHMException("Files must be bitmaps", __LINE__, __FILE__);

        cv::Mat frame_mat = cv::imread(path, CV_LOAD_IMAGE_GRAYSCALE);
        if ( frame_mat.cols != N || frame_mat.rows != N )
            throw DHMException("Images must be of size NxN", __LINE__, __FILE__);

        byte *frame = frame_mat.data;

        process_frame(frame, volume); // callback!!!

        // write volume to disk... what format? HDF5?
    }
}

void DHMProcessor::process_frame(byte *frame, float *volume) {
    // fun part goes here
    // this would be a callback
}

//}





