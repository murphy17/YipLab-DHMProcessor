#include "hip/hip_runtime.h"
/*
 * ops.cu
 *
 *  Created on: Jun 14, 2017
 *      Author: michaelmurphy
 */

#include "ops.cuh"
#include "DHMProcessor.cuh"

///////////////////////////////////////////////////////////////////////////////
// Complex arithmetic
///////////////////////////////////////////////////////////////////////////////

namespace ops {

__device__ __forceinline__ complex conj(const complex a)
{
    complex c;
    c.x = a.x;
    c.y = -a.y;
    return c;
}

__device__ __forceinline__ complex cmul(const complex a, const complex b)
{
    complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

}


///////////////////////////////////////////////////////////////////////////////
// Element-wise operations
///////////////////////////////////////////////////////////////////////////////

namespace ops {

__global__
void _b2c(const __restrict__ byte *b, complex *z)
{
    const int i = blockIdx.x;
    const int j = threadIdx.x;
    const int N = blockDim.x; // blockDim shall equal N

    z[i*N+j].x = ((float)(b[i*N+j])) / 255.f;
    z[i*N+j].y = 0.f;
}

__global__
void _freq_shift(complex *data)
{
    const int i = blockIdx.x;
    const int j = threadIdx.x;
    const int N = blockDim.x;

    const float a = (float)(1 - 2 * ((i+j) & 1));

    data[i*N+j].x *= a;
    data[i*N+j].y *= a;
}

__global__
void _modulus(const __restrict__ complex *z, float *r)
{
    const int offset = blockIdx.x * blockDim.x + threadIdx.x;
    r[offset] = hypotf(z[offset].x, z[offset].y);
}

}

///////////////////////////////////////////////////////////////////////////////
// Construct "PSF"
///////////////////////////////////////////////////////////////////////////////

namespace ops {

__global__ void _gen_filter_slice(
    complex *g,
    const float z,
    const DHMParameters p
) {
    const int i = blockIdx.x;
    const int j = threadIdx.x;

    // this is correct, 'FFT-even symmetry' - periodic extension must be symmetric about (0,0)
    float x = (i - p.N/2) * p.DX;
    float y = (j - p.N/2) * p.DY;

    float r = (-2.f / p.LAMBDA0) * norm3df(x, y, z);

    // exp(ix) = cos(x) + isin(x)
    float re, im;
    sincospif(r, &im, &re);

    // also corrects the sign flip above
    r /= -2.f * z / p.LAMBDA0;

    // re(iz) = -im(z), im(iz) = re(z)
    g[i*p.N+j] = {-im / r, re / r};
}

}

void DHMProcessor::transfer_filter_async(complex *h_filter, complex *d_filter)
{
    // generate parameters for 3D copy
    hipMemcpy3DParms q = { 0 };
    q.srcPtr.ptr = h_filter;
    q.srcPtr.pitch = (N/2+1) * sizeof(complex);
    q.srcPtr.xsize = (N/2+1);
    q.srcPtr.ysize = (N/2+1);
    q.dstPtr.ptr = d_filter;
    q.dstPtr.pitch = N * sizeof(complex);
    q.dstPtr.xsize = N;
    q.dstPtr.ysize = N;
    q.extent.width = (N/2+1) * sizeof(complex);
    q.extent.height = (N/2+1);
    q.extent.depth = NUM_SLICES;
    q.kind = hipMemcpyHostToDevice;

    CUDA_CHECK( hipMemcpy3DAsync(&q, async_stream) );
}

void DHMProcessor::gen_filter_quadrant(complex *h_filter) {
    complex *slice;
    CUDA_CHECK( hipMalloc(&slice, N*N*sizeof(complex)) );

    for (int i = 0; i < NUM_SLICES; i++)
    {
        ops::_gen_filter_slice<<<N, N>>>(slice, Z0 + i * DZ, p);
        KERNEL_CHECK();

        // FFT in-place
        CUDA_CHECK( hipfftXtExec(fft_plan, slice, slice, HIPFFT_BACKWARD) );

        // frequency shift -- eliminates need to copy later
        ops::_freq_shift<<<N, N>>>(slice);
        KERNEL_CHECK();

        // copy single quadrant to host
        CUDA_CHECK( hipMemcpy2D(
            h_filter + (N/2+1)*(N/2+1)*i,
            (N/2+1)*sizeof(complex),
            slice,
            N*sizeof(complex),
            (N/2+1)*sizeof(complex),
            N/2+1,
            hipMemcpyDeviceToHost
        ) );
    }

    CUDA_CHECK( hipFree(slice) );
}

///////////////////////////////////////////////////////////////////////////////
// Quadrant multiply kernel
///////////////////////////////////////////////////////////////////////////////

namespace ops {

// using fourfold symmetry of z
__global__
void _quad_mul(
    complex *z,
    const __restrict__ complex *w,
    const byte *mask,
    const DHMParameters p
) {
    const int i = blockIdx.x;
    const int j = threadIdx.x;
    const int ii = p.N-i;
    const int jj = p.N-j;

    if ((i>0 && i<p.N/2) && (j>0 && j<p.N/2))
    {
        complex w1 = w[i*p.N+j];
        complex w2 = w[ii*p.N+j];
        complex w3 = w[i*p.N+jj];
        complex w4 = w[ii*p.N+jj];

        for (int k = 0; k < p.NUM_SLICES; k++)
        {
            if (mask[k])
            {
                complex z_ij = z[i*p.N+j];
                z[i*p.N+j] = cmul(w1, z_ij);
                z[ii*p.N+jj] = cmul(w4, z_ij);
                z[ii*p.N+j] = cmul(w2, z_ij);
                z[i*p.N+jj] = cmul(w3, z_ij);
            }
            z += p.N*p.N;
        }
    }
    else if (i>0 && i<p.N/2)
    {
        complex w1 = w[i*p.N+j];
        complex w2 = w[ii*p.N+j];

        for (int k = 0; k < p.NUM_SLICES; k++)
        {
            if (mask[k])
            {
                complex z_ij = z[i*p.N+j];
                z[i*p.N+j] = cmul(w1, z_ij);
                z[ii*p.N+j] = cmul(w2, z_ij);
            }
            z += p.N*p.N;
        }
    }
    else if (j>0 && j<p.N/2)
    {
        complex w1 = w[i*p.N+j];
        complex w2 = w[i*p.N+jj];

        for (int k = 0; k < p.NUM_SLICES; k++)
        {
            if (mask[k])
            {
                complex z_ij = z[i*p.N+j];
                z[i*p.N+j] = cmul(w1, z_ij);
                z[i*p.N+jj] = cmul(w2, z_ij);
            }
            z += p.N*p.N;
        }
    }
    else
    {
        complex w1 = w[i*p.N+j];

        for (int k = 0; k < p.NUM_SLICES; k++)
        {
            if (mask[k])
            {
                complex z_ij = z[i*p.N+j];
                z[i*p.N+j] = cmul(w1, z_ij);
            }
            z += p.N*p.N;
        }
    }
}

}


///////////////////////////////////////////////////////////////////////////////
// Convert 3D volume to sparse (COO) format
///////////////////////////////////////////////////////////////////////////////

//typedef thrust::counting_iterator<int> IndexIterator;
//typedef thrust::device_vector<float>::iterator FloatIterator;
//typedef thrust::tuple<IndexIterator, FloatIterator> IteratorTuple;
//typedef thrust::zip_iterator<IteratorTuple> ZipIterator;

//namespace ops {
//
//struct _gen_coo_tuple : public thrust::unary_function<thrust::tuple<int, float>,float> {
//    int N;
//    __host__ __device__
//    _gen_coo_tuple(int n)
//    {
//        N = n;
//    }
//    __host__ __device__
//    COOTuple operator()(const thrust::tuple<int, float> &t)
//    {
//        int ii = thrust::get<0>(t);
//        int iz = ii / (N*N);
//        int iy = (ii - iz*N*N) / N;
//        int ix = ii - iz*N*N - iy*N;
//        float v = thrust::get<1>(t);
//        COOTuple c = {ix, iy, iz, v};
//        return c;
//    }
//};
//
//struct _filter_zeros {
//    float ZERO_THR;
//    __host__ __device__
//    _filter_zeros(float thr)
//    {
//        ZERO_THR = thr;
//    }
//    bool operator()(const thrust::tuple<int, float> t)
//    {
//        float v = thrust::get<1>(t);
//        return (v > ZERO_THR);
//    }
//};

namespace ops {

// compact rows
__global__
void _compact_rows(const float *slice, int *x, float *v, int *s, const DHMParameters p)
{
    extern __shared__ int shared_mem[];
    int *xs = shared_mem;
    float *vs = ((float *)shared_mem) + p.N;
    unsigned int *ss = ((unsigned int *)shared_mem) + 2*p.N;

    const float *row = slice + blockIdx.x * blockDim.x;

    float val = row[threadIdx.x];

    if (val > 0)
    {
        int idx = atomicInc(ss, 1);
        xs[idx] = threadIdx.x;
        vs[idx] = val;
    }

    __syncthreads();

    if (threadIdx.x < *ss)
    {
        int offset = blockIdx.x * blockDim.x + threadIdx.x;
        x[offset] = xs[threadIdx.x];
        v[offset] = vs[threadIdx.x];
    }

    if (threadIdx.x == 0)
    {
        s[blockIdx.x] = *ss;
    }
}

}

COOList DHMProcessor::rows_to_list(int *x, int *y, int z, float *v, int *s)
{
    int offset = 0;

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < s[i]; j++)
        {
            x[offset] = x[i*N+j];
            y[offset] = i;
            v[offset] = v[i*N+j];

            offset++;
        }
    }

    COOList list;

    for (int i = 0; i < offset; i++)
    {
        COOTuple t = {x[i], y[i], z, v[i]};
        list.push_back(t);
    }

    return list;
}

// not working
COOList DHMProcessor::volume_to_list(float *volume)
{
    int *d_x, *d_y, *d_s;
    float *d_v;

    int *h_x, *h_y, *h_s;
    float *h_v;

    // unified...
    CUDA_CHECK( hipMalloc(&d_x, N*N*sizeof(int)) );
    CUDA_CHECK( hipMalloc(&d_y, N*N*sizeof(int)) );
    CUDA_CHECK( hipMalloc(&d_v, N*N*sizeof(int)) );
    CUDA_CHECK( hipMalloc(&d_s, N*sizeof(int)) );

    CUDA_CHECK( hipHostMalloc(&h_x, N*N*sizeof(int)) );
    CUDA_CHECK( hipHostMalloc(&h_y, N*N*sizeof(int)) );
    CUDA_CHECK( hipHostMalloc(&h_v, N*N*sizeof(int)) );
    CUDA_CHECK( hipHostMalloc(&h_s, N*sizeof(int)) );

    std::vector<COOTuple> list;

    for (int i = 0; i < NUM_SLICES; i++)
    {
        float *slice = volume + i*N*N;
        ops::_compact_rows<<<N, N, (2*N+1)*sizeof(int)>>>(slice, d_x, d_v, d_s, p);
        KERNEL_CHECK();

        // not needed w/ unified!
        CUDA_CHECK( hipMemcpy(h_x, d_x, N*N*sizeof(int), hipMemcpyDeviceToHost) );
        CUDA_CHECK( hipMemcpy(h_y, d_y, N*N*sizeof(int), hipMemcpyDeviceToHost) );
        CUDA_CHECK( hipMemcpy(h_v, d_v, N*N*sizeof(float), hipMemcpyDeviceToHost) );
        CUDA_CHECK( hipMemcpy(h_s, d_s, N*sizeof(int), hipMemcpyDeviceToHost) );

        COOList row_list = rows_to_list(h_x, h_y, i, h_v, h_s);

        list.insert(list.end(), row_list.begin(), row_list.end());
    }

    // unified...
    CUDA_CHECK( hipFree(d_x) );
    CUDA_CHECK( hipFree(d_y) );
    CUDA_CHECK( hipFree(d_v) );
    CUDA_CHECK( hipFree(d_s) );
    CUDA_CHECK( hipFree(h_x) );
    CUDA_CHECK( hipFree(h_y) );
    CUDA_CHECK( hipFree(h_v) );
    CUDA_CHECK( hipFree(h_s) );

    return list;
}


///////////////////////////////////////////////////////////////////////////////
// I/O ops
///////////////////////////////////////////////////////////////////////////////

// should these happen in separate threads?

// load image and push to GPU
void DHMProcessor::load_image(std::string path)
{
    cv::Mat frame_mat = cv::imread(path, CV_LOAD_IMAGE_GRAYSCALE);
    if ( frame_mat.cols != N || frame_mat.rows != N ) DHM_ERROR("Images must be of size NxN");

    memcpy(h_frame, frame_mat.data, N*N*sizeof(byte));

    if (!UNIFIED_MEM)
    {
        CUDA_CHECK( hipMemcpy(d_frame, h_frame, N*N*sizeof(byte), hipMemcpyHostToDevice) );
    }
    else
    {
        CUDA_CHECK( hipHostGetDevicePointer(&d_frame, h_frame, 0) );
    }
}

// compress 3D volume to COO and save
// not obvious how to use unified mem here - Thrust allocation rules it out?
void DHMProcessor::save_volume(std::string path)
{
    COOList list = volume_to_list(d_volume);

    std::ofstream f(path, std::ios::out | std::ios::binary);
    f.write((char *)(list.data()), list.size()*sizeof(COOTuple));
    f.close();
}

void DHMProcessor::load_volume(std::string path, float *h_volume)
{
    std::ifstream f(path, std::ios::in | std::ios::binary | std::ios::ate);

    int len = f.tellg();

    COOTuple *list = new COOTuple[len];

    f.read((char *)list, len*sizeof(COOTuple));
    f.close();

    for (int i = 0; i < len; i++)
    {
        COOTuple t = list[i];
        h_volume[t.z*N*N+t.y*N+t.x] = t.v;
    }

    delete[] list;
}







